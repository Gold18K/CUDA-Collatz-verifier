﻿
// Inclusions
#include "hip/hip_runtime.h"
#include "Cuda_Utilities.h"
#include ""
#include "Kernels.cuh"

int main() {
	Cuda_Utilities::cuda_choose_device();
	Kernels::launch_collatz();
	Cuda_Utilities::cuda_device_reset();

    return 0;
}